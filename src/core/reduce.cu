#include "hip/hip_runtime.h"
#define NVCC_VISIBLE
#include <dccl/dccl.hpp>
#include "internal_common.hpp"
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

namespace dccl{

template<typename DT>
__global__ void reduce_kernel(
    const void*     sendbuf,
    void*           recvbuf,
    size_t          count,
    ncclRedOp_t     op) {
    size_t idx = blockIdx.x*blockDim.x + threadIdx.x;
    const DT*   sbuf = static_cast<const DT*>(sendbuf);
    DT*         rbuf = static_cast<DT*>(recvbuf);
    while(idx < count) {
        switch (op) {
        case ncclSum:
        case ncclAvg:
            rbuf[idx] += sbuf[idx];
            break;
        case ncclProd:
            rbuf[idx] *= rbuf[idx];
            break;
        case ncclMax:
            rbuf[idx] = (rbuf[idx]>=sbuf[idx])?rbuf[idx]:sbuf[idx];
            break;
        case ncclMin:
            rbuf[idx] = (rbuf[idx]<=sbuf[idx])?rbuf[idx]:sbuf[idx];
            break;
        default:
            break;
        }
        idx += gridDim.x*blockDim.x;
    }
}

ncclResult_t do_device_reduce(
    const void*     sendbuf,
    void*           recvbuf,
    ncclDataType_t  dtype,
    size_t          count,
    ncclRedOp_t     op,
    hipStream_t    stream) {
    // Calculate the best options for the number of blocks and block sizes.
    hipCtx_t   ctx;
    hipDevice_t    dev;
    int         num_sms;
    if (hipSuccess != cuStreamGetCtx(stream, &ctx)) {
        return ncclUnhandledCudaError;
    }
    if (hipSuccess != hipCtxPushCurrent(ctx)) {
        return ncclUnhandledCudaError;
    }
    if (hipSuccess != hipCtxGetDevice(&dev)) {
        return ncclUnhandledCudaError;
    }
    if (hipSuccess != hipDeviceGetAttribute(&num_sms,hipDeviceAttributeMultiprocessorCount,dev)) {
        return ncclUnhandledCudaError;
    }
    if (hipSuccess != hipCtxPopCurrent(&ctx)) {
        return ncclUnhandledCudaError;
    }
    const size_t nblock = num_sms;
    switch(dtype) {
    case ncclInt8:
        reduce_kernel<int8_t><<<nblock,256,0,stream>>>(sendbuf,recvbuf,count,op);
        break;
    case ncclUint8:
        reduce_kernel<uint8_t><<<nblock,256,0,stream>>>(sendbuf,recvbuf,count,op);
        break;
    case ncclInt32:
        reduce_kernel<int32_t><<<nblock,256,0,stream>>>(sendbuf,recvbuf,count,op);
        break;
    case ncclUint32:
        reduce_kernel<uint32_t><<<nblock,256,0,stream>>>(sendbuf,recvbuf,count,op);
        break;
    case ncclInt64:
        reduce_kernel<int64_t><<<nblock,256,0,stream>>>(sendbuf,recvbuf,count,op);
        break;
    case ncclUint64:
        reduce_kernel<uint64_t><<<nblock,256,0,stream>>>(sendbuf,recvbuf,count,op);
        break;
    case ncclFloat32:
        reduce_kernel<float><<<nblock,256,0,stream>>>(sendbuf,recvbuf,count,op);
        break;
    case ncclFloat64:
        reduce_kernel<double><<<nblock,256,0,stream>>>(sendbuf,recvbuf,count,op);
        break;
    case ncclFloat16:
        reduce_kernel<half><<<nblock,256,0,stream>>>(sendbuf,recvbuf,count,op);
        break;
    default:
        return ncclInvalidArgument;
    }

    return ncclSuccess;
}

}
