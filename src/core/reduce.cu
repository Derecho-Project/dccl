#include "hip/hip_runtime.h"
#define NVCC_VISIBLE
#include <dccl/dccl.hpp>
#include "internal_common.hpp"
#include <hip/hip_fp16.h>

namespace dccl{

template<typename DT>
__global__ void reduce_kernel(
    const void*     sendbuf,
    void*           recvbuf,
    size_t          count,
    ncclRedOp_t     op) {
    size_t idx = blockIdx.x*blockDim.x + threadIdx.x;
    const DT*   sbuf = static_cast<const DT*>(sendbuf);
    DT*         rbuf = static_cast<DT*>(recvbuf);
    while(idx < count) {
        switch (op) {
        case ncclSum:
        case ncclAvg:
            rbuf[idx] += sbuf[idx];
            break;
        case ncclProd:
            rbuf[idx] *= rbuf[idx];
            break;
        case ncclMax:
            rbuf[idx] = (rbuf[idx]>=sbuf[idx])?rbuf[idx]:sbuf[idx];
            break;
        case ncclMin:
            rbuf[idx] = (rbuf[idx]<=sbuf[idx])?rbuf[idx]:sbuf[idx];
            break;
        default:
            break;
        }
        idx += gridDim.x*blockDim.x;
    }
}

ncclResult_t do_device_reduce(
    const void*     sendbuf,
    void*           recvbuf,
    ncclDataType_t  dtype,
    size_t          count,
    ncclRedOp_t     op,
    hipStream_t    stream) {
    switch(dtype) {
    case ncclInt8:
        reduce_kernel<int8_t><<<1,256,0,stream>>>(sendbuf,recvbuf,count,op);
        break;
    case ncclUint8:
        reduce_kernel<uint8_t><<<1,256,0,stream>>>(sendbuf,recvbuf,count,op);
        break;
    case ncclInt32:
        reduce_kernel<int32_t><<<1,256,0,stream>>>(sendbuf,recvbuf,count,op);
        break;
    case ncclUint32:
        reduce_kernel<uint32_t><<<1,256,0,stream>>>(sendbuf,recvbuf,count,op);
        break;
    case ncclInt64:
        reduce_kernel<int64_t><<<1,256,0,stream>>>(sendbuf,recvbuf,count,op);
        break;
    case ncclUint64:
        reduce_kernel<uint64_t><<<1,256,0,stream>>>(sendbuf,recvbuf,count,op);
        break;
    case ncclFloat32:
        reduce_kernel<float><<<1,256,0,stream>>>(sendbuf,recvbuf,count,op);
        break;
    case ncclFloat64:
        reduce_kernel<double><<<1,256,0,stream>>>(sendbuf,recvbuf,count,op);
        break;
    case ncclFloat16:
        reduce_kernel<half><<<1,256,0,stream>>>(sendbuf,recvbuf,count,op);
        break;
    default:
        return ncclInvalidArgument;
    }

    return ncclSuccess;
}

}
